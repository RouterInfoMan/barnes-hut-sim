#include "hip/hip_runtime.h"
#include "cuda.hpp"

__device__ float4 fetch_utils(CUDABody_t body) {
    return make_float4(body.pos.x, body.pos.y, 1, body.mass);
}

__device__ float2 operator+(const float2 &a, const float2 &b) {
    return make_float2(a.x+b.x, a.y+b.y);
}
__device__ float2 operator+=(const float2 &a, const float2 &b) {
    return make_float2(a.x+b.x, a.y+b.y);
}
__device__ float2 operator*(const float2 &a, const float2 &b) {
    return make_float2(a.x*b.x, a.y*b.y);
}
__device__ float2 operator-(const float2 &a, const float2 &b) {
    return make_float2(a.x-b.x, a.y-b.y);
}
__device__ float2 operator/(const float2 &a, const float2 &b) {
    return make_float2(a.x/b.x, a.y/b.y);
}

__device__ float2 operator*(const float2 &a, const float &b) {
    return make_float2(a.x*b, a.y*b);
}
__device__ float2 operator/(const float2 &a, const float &b) {
    return make_float2(a.x/b, a.y/b);
}
__device__ float2 operator*(const float &a, const float2 &b) {
    return b * a;
}
__device__ float2 operator/(const float &a, const float2 &b) {
    return make_float2(a/b.x, a/b.y);
}

__device__ double norm(const float2 &a) {
    return sqrtf(a.x*a.x+a.y*a.y);
}

__device__ float2 operator-(const float2 &b) {
    return make_float2(-b.x, -b.y);
}




__device__ float2 accumulateAccel(float4 bi, float4 bj, float2 acc, float G) {
    float2 r;
    r.x = bj.x - bi.x;
    r.y = bj.y - bi.y;

    float distp = r.x * r.x + r.y * r.y + CUDA_EPSILON;
    
    float dist = sqrtf(distp);
    
    if (dist < 1.0f)
        return acc;

    float accn = G * bj.w / (dist * dist * dist);

    acc.x += r.x * accn;
    acc.y += r.y * accn; 
    return acc;
}

__global__ void updateBodies(CUDABody_t *bodies, size_t length, float G, float dt) {
    size_t true_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (true_idx >= length)
        return;
    
    float2 acc = {0, 0};
    for (size_t j = 0; j < length; j++) {
        if (true_idx != j) {
            acc = accumulateAccel(
                fetch_utils(bodies[true_idx]),
                fetch_utils(bodies[j]),
                acc,
                G
            );
        }
    }
    // if (true_idx == -1) {
    //     printf("accel=%f %f\n", acc.x, acc.y);
    //     printf("vel=%f %f\n", bodies[0].vel.x, bodies[0].vel.y);
    //     printf("pos=%f %f\n", bodies[0].pos.x, bodies[0].pos.y);
    // }
    bodies[true_idx].vel.x += acc.x * dt/2;
    bodies[true_idx].vel.y += acc.y * dt/2;

    bodies[true_idx].pos.x += bodies[true_idx].vel.x * dt;
    bodies[true_idx].pos.y += bodies[true_idx].vel.y * dt;

    bodies[true_idx].vel.x += acc.x * dt/2;
    bodies[true_idx].vel.y += acc.y * dt/2;

    // bodies[true_idx].vel.x *= 0.9999;
    // bodies[true_idx].vel.y *= 0.9999;
}

__host__ void walkCUDADevice(CUDABody_t *host_bodies, size_t length, float G, float dt) {
    CUDABody_t *bodies;

    hipMalloc(&bodies, sizeof(CUDABody_t) * length);
    hipMemcpy(bodies, host_bodies, sizeof(CUDABody_t) * length, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;
    updateBodies<<<blocksPerGrid, threadsPerBlock>>>(bodies, length, G, dt);

    hipMemcpy(host_bodies, bodies, sizeof(CUDABody_t) * length, hipMemcpyDeviceToHost);
    hipFree(bodies);
}